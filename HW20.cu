#include "hip/hip_runtime.h"
//nvcc HW20.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BODIES 6
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BODIES], Velocity[NUMBER_OF_BODIES], Force[NUMBER_OF_BODIES], Color[NUMBER_OF_BODIES];
float BodyMass[NUMBER_OF_BODIES], BodyRadius[NUMBER_OF_BODIES];
float MaxVelocity;
int Trace;
int Pause;
int PrintRate;
int PrintCount;

// Units and universal constants
double MassUnitConverter;
double LengthUnitConverter;
double TimeUnitConverter;
float GravityConstant;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitialConditions();
void drawPicture();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();
void getForces();
void updatePositions();
void nBody();
void startMeUp();
void terminalPrint();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	if(key == 'k')
	{
		Pause = 1;
		terminalPrint();
		zeroOutSystem();
		drawPicture();
		printf("\n The simulation has been zeroed out.\n");
	}
	
	if(key == '1')
	{
		float4 pos, vel;
		Pause = 1;
		terminalPrint();
		pos = centerOfMass();
		printf("\n Center of mass = (%f, %f, %f)\n", pos.x, pos.y, pos.z); 
		vel = linearVelocity();
		printf("\n Linear velocity = (%f, %f, %f)\n", vel.x, vel.y, vel.z);
	}
	
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	
	float dx = 0.05f;
	if(key == 'x')
	{
		glTranslatef(-dx, 0.0, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'X')
	{
		glTranslatef(dx, 0.0, 0.0);
		drawPicture();
	}
	
	float dy = 0.05f;
	if(key == 'y')
	{
		glTranslatef(0.0, -dy, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Y')
	{
		glTranslatef(0.0, dy, 0.0);
		drawPicture();
	}
	
	float dz = 0.05f;
	if(key == 'z')
	{
		glTranslatef(0.0, 0.0, -dz);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Z')
	{
		glTranslatef(0.0, 0.0, dz);
		drawPicture();
	}
	
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		printf("\nExiting....\n\nGood Bye\n");
		exit(0);
	}
}

void setInitialConditions()
{
	time_t t;
	float seperation;
	int test;
	int tryCount;
	float globeSize, angle1, angle2, radius;
	double massOfPolystyreneSphere;
	double diameterOfPolystyreneSphere;
	double densityOfPolystyreneSphere;
	double volumeOfPolystyreneSphere;
	double G = (8.649828e-13); //km^3/kg*hr^2
	
	// Seeding the random number generater.
	srand((unsigned) time(&t));
	
	massOfPolystyreneSphere = 0.5498; // picogram
	diameterOfPolystyreneSphere = 1.0; // micron
	densityOfPolystyreneSphere = 1.05; // g/cm^3
	volumeOfPolystyreneSphere = (PI/6.0)*diameterOfPolystyreneSphere*diameterOfPolystyreneSphere*diameterOfPolystyreneSphere;
	
	printf("\n Mass of Polystyrene Sphere = %e picograms", massOfPolystyreneSphere);
	printf("\n Diameter of Polystyrene Sphere = %e micron", diameterOfPolystyreneSphere);
	printf("\n Density of Polystyrene Sphere = %e grams/centimeters^3", densityOfPolystyreneSphere);
	printf("\n Volume of Polystyrene Sphere = %e centimeters^3", volumeOfPolystyreneSphere);
	printf("\n");
	
	MassUnitConverter = massOfPolystyreneSphere/NUMBER_OF_BODIES; // kg
	LengthUnitConverter = pow(6.0*(volumeOfPolystyreneSphere/NUMBER_OF_BODIES)/PI,1.0/3.0); // km
	TimeUnitConverter = sqrt(LengthUnitConverter*LengthUnitConverter*LengthUnitConverter/(G*MassUnitConverter)); // hr
	
	printf("\n MassUnitConverter = %e kilograms", MassUnitConverter);
	printf("\n LengthUnitConverter = %e kilometers", LengthUnitConverter);
	printf("\n TimeUnitConverter = %e hours", TimeUnitConverter);
	printf("\n");
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		BodyMass[i] /= MassUnitConverter;
		BodyRadius[i] /= LengthUnitConverter;
	}
	
	// If we did everything right the universal gravity constant should be 1.
	GravityConstant = 1.0;
	printf("\n The gravity constant = %f in our units", GravityConstant);
	
	// All spheres are the same diameter and mass so these should be 1..
	
	// Making the size of the intial globe we use to place the bodies.
	globeSize = 10.0;
	
	// You get to pick this but it is nice to print it out in common units to get a feel for what it is.
	MaxVelocity = 1.0;
	printf("\n Max velocity = %f kilometers/hour or %f miles/hour", MaxVelocity*LengthUnitConverter/TimeUnitConverter, (MaxVelocity*LengthUnitConverter/TimeUnitConverter)*0.621371);
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// Setting the balls randomly in a large sphere and not letting them be right on top of each other.
		test = 0;
		tryCount = 0;
		while(test == 0)
		{
			// Get random position.
			angle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			angle2 = ((float)rand()/(float)RAND_MAX)*PI;
			radius = ((float)rand()/(float)RAND_MAX)*globeSize;
			Position[i].x = radius*cos(angle1)*sin(angle2);
			Position[i].y = radius*sin(angle1)*sin(angle2);
			Position[i].z = radius*cos(angle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < (BodyRadius[i] + BodyRadius[j]))
				{
					test = 0;
					break;
				}
			}
			tryCount++;
			if(1000 < tryCount)
			{
				printf("\n\n We tried 1000 times to set the position of body %d unsuccessfully.",i);
				printf("\n Something is wrong with your setup.");
				printf("\n Good Bye. \n");
				exit(0);
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		Velocity[i].x = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		
		// Color of each asteroid. 
		Color[i].x = 0.35;
		Color[i].y = 0.22;
		Color[i].z = 0.16;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	// Making it run for 10 days.
	// Taking days to hours then to our units.
	TotalRunTime = 10.0*24.0/TimeUnitConverter;
	RunTime = 0.0;
	Dt = 0.001;
	// How many time steps between terminal prints
	PrintRate = 10;
	
	printf("\n");
	printf("\n Initial Conditions Have Been Set.");
	printf("\n");
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
		glTranslatef(Position[i].x, Position[i].y, Position[i].z);
		glutSolidSphere(BodyRadius[i], 30, 30);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
    centerOfMass.x += Position[i].x*BodyMass[i];
	  centerOfMass.y += Position[i].y*BodyMass[i];
	  centerOfMass.z += Position[i].z*BodyMass[i];
		totalMass += BodyMass[i];
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
    linearVelocity.x += Velocity[i].x*BodyMass[i];
		linearVelocity.y += Velocity[i].y*BodyMass[i];
		linearVelocity.z += Velocity[i].z*BodyMass[i];
		totalMass += BodyMass[i];
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		Position[i].x -= pos.x;
		Position[i].y -= pos.y;
		Position[i].z -= pos.z;
		
		Velocity[i].x -= vel.x;
		Velocity[i].y -= vel.y;
		Velocity[i].z -= vel.z;
	}
}

void getForces()
{
	float inOut;
	float kSphere,kSphereReduction;
	float4 d, unit, dv;
	float magnitude;
	float intersectionArea; 
	float epsilon = 0.01;
	float r1,r2,temp;
	
	// Zeroing forces outside of the force loop just to be safe.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	kSphere = 1000.0;
	kSphereReduction = 0.3;
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{	
		// This adds forces between asteriods.
		for(int j = 0; j < i; j++)
		{
			d.x = Position[j].x - Position[i].x;
			d.y = Position[j].y - Position[i].y;
			d.z = Position[j].z - Position[i].z;
			d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
			unit.x = d.x/d.w;
			unit.y = d.y/d.w;
			unit.z = d.z/d.w;
			
			// Nonelastic sphere collisions 
			if(d.w < (BodyRadius[i] + BodyRadius[j]))
			{
				// If the seperation gets too small the sphers may go through each other.
				// If you are ok with that you do not need this if statement.
				if(d.w < epsilon)
				{
					printf("\n Spheres %d and %d got to close. Make your sphere repultion stronger\n", i, j);
					exit(0);
				}
				
				// Finding which body is largest.
				if(BodyRadius[j] < BodyRadius[i])
				{
					r1 = BodyRadius[i];
					r2 = BodyRadius[j];
				}
				else
				{
					r1 = BodyRadius[j];
					r2 = BodyRadius[i];
				} 

				// Finding the intersection area.
				// The intersection area gets too large (If one sphere goes into the other).
				// Set it as the radius of the smaller sphere.
				temp = (d.w*d.w - r1*r1 + r2*r2 )/(2*d.w); 
				if(0.0 < temp)
				{
					intersectionArea = PI*(r2*r2 - temp*temp);
				}
				else
				{
					intersectionArea = PI*(r2*r2);
				}
				
				dv.x = Velocity[j].x - Velocity[i].x;
				dv.y = Velocity[j].y - Velocity[i].y;
				dv.z = Velocity[j].z - Velocity[i].z;
				inOut = d.x*dv.x + d.y*dv.y + d.z*dv.z;
				if(inOut < 0.0) magnitude = kSphere*intersectionArea; // If inOut is negative the sphere are converging.
				else magnitude = kSphereReduction*kSphere*intersectionArea; // If inOut is positive the sphere are diverging.
				
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*unit.x;
				Force[i].y -= magnitude*unit.y;
				Force[i].z -= magnitude*unit.z;
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*unit.x;
				Force[j].y += magnitude*unit.y;
				Force[j].z += magnitude*unit.z;
				
				// This adds the gravity between asteroids but the gravity is lock in at what it 
				// was at impact.
				magnitude = GravityConstant*BodyMass[i]*BodyMass[j]/((BodyRadius[i] + BodyRadius[j])*(BodyRadius[i] + BodyRadius[j]));
				Force[i].x += magnitude*unit.x;
				Force[i].y += magnitude*unit.y;
				Force[i].z += magnitude*unit.z;
				
				Force[j].x -= magnitude*unit.x;
				Force[j].y -= magnitude*unit.y;
				Force[j].z -= magnitude*unit.z;
			}
			else
			{
				// This adds the gravity between asteroids when they are not touching.
				magnitude = GravityConstant*BodyMass[i]*BodyMass[j]/(d.w*d.w);
				Force[i].x += magnitude*unit.x;
				Force[i].y += magnitude*unit.y;
				Force[i].z += magnitude*unit.z;
				
				Force[j].x -= magnitude*unit.x;
				Force[j].y -= magnitude*unit.y;
				Force[j].z -= magnitude*unit.z;
			}
		}
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/BodyMass[i])*(Dt/2.0);
			Velocity[i].y += (Force[i].y/BodyMass[i])*(Dt/2.0);
			Velocity[i].z += (Force[i].z/BodyMass[i])*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/BodyMass[i])*Dt;
			Velocity[i].y += (Force[i].y/BodyMass[i])*Dt;
			Velocity[i].z += (Force[i].z/BodyMass[i])*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	
	RunTime += Dt;
	PrintCount++;
	
	if(PrintCount == PrintRate)
	{
		terminalPrint();
		PrintCount = 0;
	}
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	PrintCount = 0;
	setInitialConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n");
	printf("\n X/x: Move Right move left");
	printf("\n Y/y: Move Up move down");
	printf("\n Z/z: Move in move out");
	
	printf("\n");
	printf("\n k: Will zero out the center of mass and linear velocity of the system.");
	printf("\n 1: Will print the center of mass and the linear velocity of the system.");
	printf("\n");
	printf("\033[0m");
	printf("\n t: Trace on/off toggle --> ");
	printf(" Tracing is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;32mON\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mOFF\n" "\e[m");
	}
	
	printf("\033[0m");
	printf(" p: pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf(" q: Terminates the simulation");
	
	// Print the time out in hours.
	printf("\n\n Time = %f \033[0;34mhours", RunTime*TimeUnitConverter);
	printf("\033[0m");
	printf("\n");
}


int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 50.0;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 25.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
