#include "hip/hip_runtime.h"
//nvcc WallBroken.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BALLS 100
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BALLS], Velocity[NUMBER_OF_BALLS], Force[NUMBER_OF_BALLS], Color[NUMBER_OF_BALLS];
float SphereMass;
float SphereDiameter;
float MaxVelocity;
int Trace;
int Pause;
int PrintRate;
int PrintCount;

// Units and universal constants
float MassUnitConverter;
float LengthUnitConverter;
float TimeUnitConverter;
float GravityConstant;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitailConditions();
void drawPicture();
float4 centerOfMass();
float4 linearVelocity();
void getForces();
void updatePositions();
void nBody();
void startMeUp();
void terminalPrint();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	if(key == 'k')
	{
		//float4 pos, vel;
		//Pause = 1;
		//terminalPrint();
		// ??????????????????????????????????????????
		// Zero out center of mass and linear velocity of the system.
		//drawPicture();
		printf("\n The simulation has been zeroed out.\n");
	}
	
	if(key == '1')
	{
		//float4 pos, vel;
		//Pause = 1;
		//terminalPrint();
		// ??????????????????????????????????????????
		//Print out center of mass and linear velocity of the system.
	}
	
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	
	float dx = 0.05f;
	if(key == 'x')
	{
		glTranslatef(-dx, 0.0, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'X')
	{
		glTranslatef(dx, 0.0, 0.0);
		drawPicture();
	}
	
	float dy = 0.05f;
	if(key == 'y')
	{
		glTranslatef(0.0, -dy, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Y')
	{
		glTranslatef(0.0, dy, 0.0);
		drawPicture();
	}
	
	float dz = 0.05f;
	if(key == 'z')
	{
		glTranslatef(0.0, 0.0, -dz);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Z')
	{
		glTranslatef(0.0, 0.0, dz);
		drawPicture();
	}
	
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		printf("\nExiting....\n\nGood Bye\n");
		exit(0);
	}
}

void setInitailConditions()
{
	time_t t;
	float seperation;
	int test;
	float maxSphereSize, angle1, angle2, radius;
	
	// Seeding the random number generater.
	srand((unsigned) time(&t));
	
	// The units that we will use to contect us to the outside world are: 
	// kilometers (km)
	// kilograms (kg)
	// hours (hr)
	// If you multiply one of our units by this number it will convert it the outside world units.
	// If you divide an outside world unit by this number it will convert it to our units
	// We are setting the mass unit to be the mass of Ceres.
	// We are settting the length unit to be th diameter of Ceres.
	// We are setting the time unit to be the such that the universal gravity constant is 1.
	MassUnitConverter = 9.383e20; // kg
	LengthUnitConverter = 940.0; // km
	TimeUnitConverter = 3642.0/(60.0*60.0); // hr
	printf("\n MassUnitConverter = %e kilograms", MassUnitConverter);
	printf("\n LengthUnitConverter = %e kilometers", LengthUnitConverter);
	printf("\n TimeUnitConverter = %e hours", TimeUnitConverter);
	
	// If we did everthing right the universal gravity constant should be 1.
	GravityConstant = 1.0;
	printf("\n The gravity constant = %f in our units", GravityConstant);
	
	// All spheres are the same diameter and mass of Ceres so these should be 1..
	SphereDiameter = 1.0;
	SphereMass = 1.0;
	
	// Making the size of the intial sphere I out the shpers in 50 times bigger than a sphere.
	maxSphereSize = 10.0*SphereDiameter;
	
	// You get to pick this but it is nice to print it out in common units to get a feel for what it is.
	MaxVelocity = 1.0;
	printf("\n Max velocity = %f kilometers/hour or %f miles/hour", MaxVelocity*LengthUnitConverter/TimeUnitConverter, (MaxVelocity*LengthUnitConverter/TimeUnitConverter)*0.621371);
	
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// Settting the balls randomly in a large sphere and not letting them be right on top of each other.
		test = 0;
		while(test == 0)
		{
			// Get random position.
			angle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			angle2 = ((float)rand()/(float)RAND_MAX)*PI;
			radius = ((float)rand()/(float)RAND_MAX)*maxSphereSize;
			Position[i].x = radius*cos(angle1)*sin(angle2);
			Position[i].y = radius*sin(angle1)*sin(angle2);
			Position[i].z = radius*cos(angle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < SphereDiameter)
				{
					test = 0;
					break;
				}
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		Velocity[i].x = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		
		// Color of each asteroid. 
		Color[i].x = 0.35;
		Color[i].y = 0.22;
		Color[i].z = 0.16;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	// Making it run for 10 days.
	// Taking days to hours then to our units.
	TotalRunTime = 10.0*24.0/TimeUnitConverter;
	RunTime = 0.0;
	Dt = 0.001;
	// How many time steps between termenal prints
	PrintRate = 10;
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(SphereDiameter/2.0, 30, 30);
		glPopMatrix();
	}
	
	// ???????????????????????????????????????????????
	// Draw a cool 10X10 wall centered at (25,0,0) perpendicular to the x axis.
	
	glutSwapBuffers();
}

float4 centerOfMass()
{
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	
	// ????????????????????????????????????????????????????????
	// Return the center of mass of the system.

	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	
	// ????????????????????????????????????????????????????????
	// Return the linear velocity of the system.
	
	return(linearVelocity);
}

void getForces()
{
	float inOut;
	float kSphereReduction = 0.5;
	float dvx, dvy, dvz;
	float kSphere;
	float sphereRadius = SphereDiameter/2.0;
	float d, dx, dy, dz;
	float magnitude;
	
	// Zeroing forces outside of the force loop just to be safe.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	kSphere = 1000.0;
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{	
		for(int j = 0; j < i; j++)
		{
			dx = Position[j].x - Position[i].x;
			dy = Position[j].y - Position[i].y;
			dz = Position[j].z - Position[i].z;
			d = sqrt(dx*dx + dy*dy + dz*dz);
			
			// Nonelastic sphere collisions 
			if(d < SphereDiameter)
			{
				// If the seperation gets smaller than a radius something is wrong.
				if(d < sphereRadius)
				{
					printf("\n Spheres %d and %d got to close. Make your sphere repultion stronger\n", i, j);
					exit(0);
				}
				
				dvx = Velocity[j].x - Velocity[i].x;
				dvy = Velocity[j].y - Velocity[i].y;
				dvz = Velocity[j].z - Velocity[i].z;
				inOut = dx*dvx + dy*dvy + dz*dvz;
				if(inOut < 0.0) magnitude = kSphere*(SphereDiameter - d); // If inOut is negative the sphere are converging.
				else magnitude = kSphereReduction*kSphere*(SphereDiameter - d); // If inOut is positive the sphere are diverging.
				
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*(dx/d);
				Force[i].y -= magnitude*(dy/d);
				Force[i].z -= magnitude*(dz/d);
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*(dx/d);
				Force[j].y += magnitude*(dy/d);
				Force[j].z += magnitude*(dz/d);
				
				// This adds the gravity between asteroids but the gravity is lock it at what it 
				// was at impact.
				magnitude = GravityConstant*SphereMass*SphereMass/(SphereDiameter*SphereDiameter);
				Force[i].x += magnitude*(dx/d);
				Force[i].y += magnitude*(dy/d);
				Force[i].z += magnitude*(dz/d);
				
				Force[j].x -= magnitude*(dx/d);
				Force[j].y -= magnitude*(dy/d);
				Force[j].z -= magnitude*(dz/d);
			}
			else
			{
				// This adds the gravity between asteroids.
				magnitude = GravityConstant*SphereMass*SphereMass/(d*d);
				Force[i].x += magnitude*(dx/d);
				Force[i].y += magnitude*(dy/d);
				Force[i].z += magnitude*(dz/d);
				
				Force[j].x -= magnitude*(dx/d);
				Force[j].y -= magnitude*(dy/d);
				Force[j].z -= magnitude*(dz/d);
			}
		}
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/SphereMass)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/SphereMass)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/SphereMass)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/SphereMass)*Dt;
			Velocity[i].y += (Force[i].y/SphereMass)*Dt;
			Velocity[i].z += (Force[i].z/SphereMass)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
	// ???????????????????????????????????????????????????????????
	// Quantum Chuck Norris is always in a superposition, and he doesn't care if you observe him or not. 
	// And don't even think about trying to entangle him, because he's spooky both up close and at a distance.
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	
	RunTime += Dt;
	PrintCount++;
	
	if(PrintCount == PrintRate)
	{
		terminalPrint();
		PrintCount = 0;
	}
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	PrintCount = 0;
	setInitailConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n");
	printf("\n X/x: Move Right move left");
	printf("\n Y/y: Move Up move down");
	printf("\n Z/z: Move in move out");
	
	printf("\n");
	printf("\n k: Will zero out the center of mass and linear velocity of the system.");
	printf("\n 1: Will print the center of mass and the linear velocity of the system.");
	
	printf("\033[0m");
	printf("\n t: Trace on/off toggle --> ");
	printf(" Tracing is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;32mON\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mOFF\n" "\e[m");
	}
	
	printf("\033[0m");
	printf(" p: pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf(" q: Terminates the simulation");
	
	// Print the time out in hours.
	printf("\n\n Time = %f \033[0;34mhours", RunTime*TimeUnitConverter);
	printf("\033[0m");
	printf("\n");
}


int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 50.0*SphereDiameter;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 25.0*SphereDiameter;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
