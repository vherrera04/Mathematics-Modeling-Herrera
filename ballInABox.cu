#include "hip/hip_runtime.h"
//nvcc ballInABox.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position, Velocity, Force;
float SphereMass;
float SphereDiameter;
float BoxSideLength;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

void setInitailConditions();
void drawPicture();
void getForces();
void updatePositions();
void nBody();
void startMeUp();

void Display()
{
	drawPicture();
}

void idle()
{
	nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void setInitailConditions()
{
	Position.x = 0.0;
	Position.y = 0.0;
	Position.z = 0.0;
	
	Velocity.x = 0.5;
	Velocity.y = 0.5;
	Velocity.z = 0.5;
	
	Force.x = 0.0;
	Force.y = 0.0;
	Force.z = 0.0;
	
	SphereDiameter = 0.5;
	
	SphereMass = 1.0;
	
	BoxSideLength = 5.0;
	
	TotalRunTime = 10000.0;
	RunTime = 0.0;
	Dt = 0.001;
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	float halfSide = BoxSideLength/2.0;
	
	glColor3d(1.0, 1.0, 0.0);
	glPushMatrix();
		glTranslatef(Position.x, Position.y, Position.z);
		glutSolidSphere(SphereDiameter/2.0, 30, 30);
	glPopMatrix();
	
	glLineWidth(3.0);
	//Drawing front of box
	glColor3d(0.0, 1.0, 0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
	glEnd();
	//Drawing back of box
	glColor3d(1.0, 1.0, 1.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off right side
	glBegin(GL_LINES);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off left side
	glBegin(GL_LINES);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	
	
	glutSwapBuffers();
}

void getForces()
{
	// !!!! you probably should work on these guys/gals.
	Force.x = 0.0;
	Force.y = 0.0;
	Force.z = 0.0;
}

void updatePositions()
{
	// These are the LeapFrog formulas.
	if(RunTime == 0.0)
	{
		Velocity.x += (Force.x/SphereMass)*(Dt/2.0);
		Velocity.y += (Force.y/SphereMass)*(Dt/2.0);
		Velocity.z += (Force.z/SphereMass)*(Dt/2.0);
	}
	else
	{
		Velocity.x += (Force.x/SphereMass)*Dt;
		Velocity.y += (Force.y/SphereMass)*Dt;
		Velocity.z += (Force.z/SphereMass)*Dt;
	}

	Position.x += Velocity.x*Dt;
	Position.y += Velocity.y*Dt;
	Position.z += Velocity.z*Dt;
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	printf("\n Time = %f", RunTime);
	RunTime += Dt;
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	setInitailConditions();
}

int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	//Far = BoxSideLength;
	Far = 10.0;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 6.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	//glutMouseFunc(mymouse);
	//glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
