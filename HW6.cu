#include "hip/hip_runtime.h"
//nvcc HW6.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BALLS 20
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BALLS], Velocity[NUMBER_OF_BALLS], Force[NUMBER_OF_BALLS], Color[NUMBER_OF_BALLS];
float SphereMass;
float SphereDiameter;
float BoxSideLength;
float MaxVelocity;
int Trace;
int Pause;
int PrintRate;
int PrintCount;

// Units and universal constants
float MassUnitConverter;
float LengthUnitConverter;
float TimeUnitConverter;
float GravityConstant;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitialConditions();
void drawPicture();
void getForces();
void updatePositions();
void nBody();
void startMeUp();
void terminalPrint();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	
	// ?????????????????????????????????????????????????????????????
	// Add left, right, up, and down functionality to your simulation.
	float dx = 0.05f;
    	float dy = 0.05f;
	float dz = 0.05f;

	if(key == 'z')
	{
		glTranslatef(0.0, 0.0, -dz);
		drawPicture();
		terminalPrint();
	}

	if(key == 'Z')
	{
		glTranslatef(0.0, 0.0, dz);
		drawPicture();
	}

	if(key == 'a') // Move left
   	 {
        	glTranslatef(dx, 0.0, 0.0);
        	drawPicture();
    	}

    	if(key == 'd') // Move right
    	{
        	glTranslatef(-dx, 0.0, 0.0);
        	drawPicture();
    	}

    	if(key == 'w') // Move up
    	{
        	glTranslatef(0.0, -dy, 0.0);
        	drawPicture();
    	}

    	if(key == 's') // Move down
    	{
        	glTranslatef(0.0, dy, 0.0);
        	drawPicture();
    	}
    
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		printf("\nExiting....\n\nGood Bye\n");
		exit(0);
	}
}

void setInitialConditions()
{
	time_t t;
	float randomNumber;
	//float sphereRadius;
	float seperation;
	int test;
	
	// Seeding the random number generater.
	srand((unsigned) time(&t));
	
	// The units that we will use to contect us to the outside world are: 
	// kilometers (km)
	// kilograms (kg)
	// hours (hr)
	// If you multiply one of our units by this number it will convert it the outside world units.
	// If you divide an outside world unit by this number it will convert it to our units
	// We are setting the mass unit to be the mass of Ceres.
	// We are settting the length unit to be th diameter of Ceres.
	// We are setting the time unit to be the such that the universal gravity constant is 1.
	MassUnitConverter = 9.383e20; // kg
	LengthUnitConverter = 940.0; // km
	TimeUnitConverter = 3642.0/(60.0*60.0); // hr
	printf("\n MassUnitConverter = %e kilograms", MassUnitConverter);
	printf("\n LengthUnitConverter = %e kilometers", LengthUnitConverter);
	printf("\n TimeUnitConverter = %e hours", TimeUnitConverter);
	
	// If we did everthing right the universal gravity constant should be 1.
	GravityConstant = 1.0;
	printf("\n The gravity constant = %f in our units", GravityConstant);
	
	// All spheres are the same diameter and mass of Ceres so these should be 1..
	SphereDiameter = 1.0;
	SphereMass = 1.0;
	sphereRadius = SphereDiameter/2.0;
	
	// You get to pick this but it is nice to print it out in common units to get a feel for what it is.
	MaxVelocity = 20.0;
	printf("\n Max velocity = %f kilometers/hour or %f miles/hour", MaxVelocity*LengthUnitConverter/TimeUnitConverter, (MaxVelocity*LengthUnitConverter/TimeUnitConverter)*0.621371);
	
	// ??????????????????????????????????????????????????
	// Take the asteroids out of the box so you will not need these. Also remove them from the set of global and local variables 
	
	// You will be initially putting the asteroids inside a big sphere 
	// so you will need a local variable call it maxSphereSize and two other local variables
	// call them angle1 and angle2.
	float maxSphereSize = 10.0; // Size of the sphere within which particles will be placed
    	float angle1, angle2;
	
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// Setting the balls randomly in a large sphere and not letting them be right on top of each other.
		test = 0;
		while(test == 0)
		{
			// ?????????????????????????????????????????????
			// Change this from a box to a sphere.
			// Get random position.

			angle1 = ((float)rand() / (float)RAND_MAX) * 2.0f * PI;
            		angle2 = ((float)rand() / (float)RAND_MAX) * PI;
            		float r = ((float)rand() / (float)RAND_MAX) * maxSphereSize;
            
           		Position[i].x = r * sin(angle2) * cos(angle1);
            		Position[i].y = r * sin(angle2) * sin(angle1);
            		Position[i].z = r * cos(angle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < SphereDiameter)
				{
					test = 0;
					break;
				}
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].x = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = randomNumber;
		
		// Color of each asteroid. 
		Color[i].x = 0.35;
		Color[i].y = 0.22;
		Color[i].z = 0.16;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	// Making it run for 10 days.
	// Taking days to hours then to our units.
	TotalRunTime = 10.0*24.0/TimeUnitConverter;
	RunTime = 0.0;
	Dt = 0.001;
	// How many time steps between termenal prints
	PrintRate = 10;
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	float halfSide = BoxSideLength/2.0;
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(SphereDiameter/2.0, 30, 30);
		glPopMatrix();
	}
	
	// ????????????????????????????????????????????????????????
	// If the asteroids are not going to live in a box why draw it. 
	
	glutSwapBuffers();
}

void getForces()
{
	// ????????????????????????????????????????????
	// We aren't going to have walls in our new world so you will not need these.
	
	
	// ????????????????????????????????????????????
	// These are a new variable you will use when making the asteroids collide inelastically. 
	float inOut;
	float kSphereReduction;
	float dvx, dvy, dvz;
	
	float kSphere;
	float sphereRadius = SphereDiameter/2.0;
	float d, dx, dy, dz;
	float magnitude;
	
	// Zeroing forces outside of the force loop just to be safe.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	kSphere = 1000.0;
	kSphereReduction = 0.5; // value for inelastic collision reduction
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{	
		// ???????????????????????????????????????????????????????????????????
		// Asteroids are free spirits. You can't keep them in a box. 
		// Take them out of the box and let them run free, as they were meant to live!
		
		
		for(int j = 0; j < i; j++)
		{
			dx = Position[j].x - Position[i].x;
			dy = Position[j].y - Position[i].y;
			dz = Position[j].z - Position[i].z;
			d = sqrt(dx*dx + dy*dy + dz*dz);
			
			// ?????????????????????????????????????????????????????
			// This causes the asteroids to bounce off of each other elastically.
			// Make this a nonelastic bounce.
			// Make two local variable inOut and kSphereReduction and fix this problem.
			// You will also need local variables dvx, dvy, dvz.
			// Also check and see if the seperation is less than the radius.
			// If it is print out a note to make your repultion stronger and termenate the program.
			if(d < SphereDiameter)
			{
				// ?????????????????
				// I did the radius check for you.
				if(d < sphereRadius)
				{
					printf("\n Spheres %d and %d got to close. Make your sphere repultion stronger\n", i, j);
					exit(0);
				}

				inOut = SphereDiameter - d; //calculates the overlap between two spheres

 				// Compute repulsive force magnitude with reduction for inelastic collisions
				magnitude = kSphere*inOut*kSphereReduction;
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*(dx/d);
				Force[i].y -= magnitude*(dy/d);
				Force[i].z -= magnitude*(dz/d);
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*(dx/d);
				Force[j].y += magnitude*(dy/d);
				Force[j].z += magnitude*(dz/d);

				//Relative Velocity
				dvx = Velocity[j].x - Velocity[i].x;
                		dvy = Velocity[j].y - Velocity[i].y;
                		dvz = Velocity[j].z - Velocity[i].z;
                
                		float dotProduct = dx * dvx + dy * dvy + dz * dvz; //helps determine amount of energy being exchanged during collision
                		float restitutionCoefficient = 0.2; // value for inelastic collision

                		if (dotProduct > 0)
                		{
                    			float collisionMagnitude = (1 + restitutionCoefficient) * dotProduct / (1 / SphereMass + 1 / SphereMass); //calculates the magnitude of the force adjustment due to the collision and takes into account how much kinetic energy is lost in an inelastic collision.

                    			Force[i].x -= collisionMagnitude * (dx / d);
                    			Force[i].y -= collisionMagnitude * (dy / d);
                    			Force[i].z -= collisionMagnitude * (dz / d);

                    			Force[j].x += collisionMagnitude * (dx / d);
                    			Force[j].y += collisionMagnitude * (dy / d);
                    			Force[j].z += collisionMagnitude * (dz / d);
				}
			}
			
			// This adds the gravity between asteroids.
			magnitude = GravityConstant*SphereMass*SphereMass/(d*d);
			Force[i].x += magnitude*(dx/d);
			Force[i].y += magnitude*(dy/d);
			Force[i].z += magnitude*(dz/d);
			
			Force[j].x -= magnitude*(dx/d);
			Force[j].y -= magnitude*(dy/d);
			Force[j].z -= magnitude*(dz/d);
			
			// ???????????? Nothing to do. Just a new comic relief.
			// A lady walks into a bar, throws her credit card down, and says, 
			// 'Give me a beer, then half a beer, then a quarter of a beer, then an eighth of a beer, 
			// and just keep them coming.' The bartender pours the lady two beers and says, 
			// 'In this business, you have to know your customer's limits.'
			//
			// Then Chuck Norris's wife walks in, throws her credit card down, and says, 
			// 'Give me a beer, then half a beer, then a third of a beer, then a quarter of a beer, 
			// then a fifth of a beer, and just keep them coming. And when you're done with that, 
			// give me a whiskey chaser.'
			//
			// If you're not laughing, ask Dr. Crawford to explain it in his analysis class. Or ask Kyle.
		}
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/SphereMass)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/SphereMass)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/SphereMass)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/SphereMass)*Dt;
			Velocity[i].y += (Force[i].y/SphereMass)*Dt;
			Velocity[i].z += (Force[i].z/SphereMass)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	
	RunTime += Dt;
	PrintCount++;
	
	if(PrintCount == PrintRate)
	{
		terminalPrint();
		PrintCount = 0;
	}
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	PrintCount = 0;
	setInitialConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	// ????????????????????????????????????????
	// let people know how to move left, right, up, and down.
	printf("\n");
	printf("\n Z/z: Move in/move out");
	printf("\n a: Move left");
    	printf("\n d: Move right");
    	printf("\n w: Move up");
    	printf("\n s: Move down");
	
	printf("\033[0m");
	printf("\n t: Trace on/off toggle --> ");
	printf(" Tracing is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;32mON\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mOFF\n" "\e[m");
	}
	
	printf("\033[0m");
	printf(" p: pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf(" q: Terminates the simulation");
	
	// Print the time out in hours.
	printf("\n\n Time = %f \033[0;34mhours", RunTime/TimeUnitConverter);
	printf("\033[0m");
	printf("\n");
}


int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 50.0*SphereDiameter;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 15.0*SphereDiameter;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
