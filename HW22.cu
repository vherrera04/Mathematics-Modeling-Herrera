#include "hip/hip_runtime.h"
// gcc HW22.c -o tower -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 6

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define PRINT 100
#define DAMP 0.3

#define G 1.0

#define DT 0.001

#define EYE 5.0
#define FAR 50.0

#define STOP_TIME 100.0

#define FLOOR_STRENGTH 200.0
#define SHERE_RADIUS 0.2
#define DROP_HIEGHT 5.0

// Globals
float Px[N], Py[N], Pz[N];
float Vx[N], Vy[N], Vz[N];
float Fx[N], Fy[N], Fz[N];
float Mass[N], CompressionStrength[N][N], TensionStrength[N][N], NaturalLength[N][N]; 
float Red[N][N], Green[N][N], Blue[N][N];

void set_initial_conditions()
{
	int i,j;
	
	//Zeroing all matrices
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			CompressionStrength[i][j] = 0.0;
			TensionStrength[i][j] = 0.0;
			NaturalLength[i][j] = 0.0;
			
			Red[i][j] = 0.0;
			Green[i][j] = 0.0;
			Blue[i][j] = 0.0;
		}
	}
	
	//Setting node masses
	for(i = 0; i < N; i++)
	{	
		Mass[i] = 1.0;
	}
	
	//Setting node velocities
	for(i = 0; i < N; i++)
	{	
		Vx[i] = 0.0;
		Vy[i] = 0.0;
		Vz[i] = 0.0;
	}
	
	//Setting connector attributes (most of the matrix is wasted)
	CompressionStrength[0][1] = 10.0;
	TensionStrength[0][1] = 10.0;
	NaturalLength[0][1] = 2.0;
	
	CompressionStrength[0][2] = 10.0;
	TensionStrength[0][2] = 10.0;
	NaturalLength[0][2] = 2.0;
	
	CompressionStrength[0][3] = 10.0;
	TensionStrength[0][3] = 10.0;
	NaturalLength[0][3] = 2.0;
	
	CompressionStrength[1][2] = 10.0;
	TensionStrength[1][2] = 10.0;
	NaturalLength[1][2] = 2.0;
	
	CompressionStrength[1][3] = 10.0;
	TensionStrength[1][3] = 10.0;
	NaturalLength[1][3] = 2.0;
	
	CompressionStrength[2][3] = 10.0;
	TensionStrength[2][3] = 10.0;
	NaturalLength[2][3] = 2.0;

	CompressionStrength[4][0] = 10.0;
	TensionStrength[4][0] = 10.0;
	NaturalLength[4][0] = 2.0;

	CompressionStrength[4][1] = 10.0;
	TensionStrength[4][1] = 10.0;
	NaturalLength[4][1] = 2.0;

	CompressionStrength[4][2] = 10.0;
	TensionStrength[4][2] = 10.0;
	NaturalLength[4][2] = 2.0;

	CompressionStrength[5][1] = 10.0;
	TensionStrength[5][1] = 10.0;
	NaturalLength[5][1] = 2.0;

	CompressionStrength[5][2] = 10.0;
	TensionStrength[5][2] = 10.0;
	NaturalLength[5][2] = 2.0;

	CompressionStrength[5][3] = 10.0;
	TensionStrength[5][3] = 10.0;
	NaturalLength[5][3] = 2.0;
	
	//Setting node positions
	Px[0] = 0.0;
	Py[0] = 0.0 + DROP_HIEGHT;
	Pz[0] = 1.0;
	
	Px[1] = 1.0;
	Py[1] = 0.0 + DROP_HIEGHT;
	Pz[1] = 0.0;
	
	Px[2] = -1.0;
	Py[2] = 0.0 + DROP_HIEGHT;
	Pz[2] = 0.0;
	
	Px[3] = 0.0;
	Py[3] = 1.0 + DROP_HIEGHT;
	Pz[3] = 0.0;

	Px[4] = 0.0;
	Py[4] = -1.0 + DROP_HIEGHT;
	Pz[4] = 0.0;

	Px[5] = 0.0;
	Py[5] = 0.0 + DROP_HIEGHT;
	Pz[5] = -1.0;
}

void draw_picture()
{
	int i;
	
	//Clearing the picture
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	//Drawing the nodes
	for(i = 0; i < N; i++)
	{
		if(i == 0) glColor3d(1.0,1.0,1.0);
		if(i == 1) glColor3d(0.0,1.0,0.0);
		if(i == 2) glColor3d(1.0,0.0,0.0);
		if(i == 3) glColor3d(1.0,0.0,1.0);
		if(i == 4) glColor3d(0.0, 0.5, 1.0); 
    		if(i == 5) glColor3d(1.0, 1.0, 0.0);
		glPushMatrix();
		glTranslatef(Px[i], Py[i], Pz[i]);
		glutSolidSphere(SHERE_RADIUS,20,20);
		glPopMatrix();
	}
	
	//Drawing the Connectors (red if compressed, blue is stretched)
	glLineWidth(8.0);
	glColor3d(Red[0][1],Green[0][1],Blue[0][1]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[0], Py[0], Pz[0]);  
		glVertex3f(Px[1], Py[1], Pz[1]);   
	glEnd();
	glColor3d(Red[0][2],Green[0][2],Blue[0][2]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[0], Py[0], Pz[0]);   
		glVertex3f(Px[2], Py[2], Pz[2]); 
	glEnd();
	glColor3d(Red[0][3],Green[0][3],Blue[0][3]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[0], Py[0], Pz[0]);   
		glVertex3f(Px[3], Py[3], Pz[3]); 
	glEnd();
	glColor3d(Red[1][2],Green[1][2],Blue[1][2]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[1], Py[1], Pz[1]);   
		glVertex3f(Px[2], Py[2], Pz[2]); 
	glEnd();
	glColor3d(Red[1][3],Green[1][3],Blue[1][3]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[1], Py[1], Pz[1]);   
		glVertex3f(Px[3], Py[3], Pz[3]); 
	glEnd();
	glColor3d(Red[2][3],Green[2][3],Blue[2][3]);
	glBegin(GL_LINE_STRIP);
		glVertex3f(Px[2], Py[2], Pz[2]);   
		glVertex3f(Px[3], Py[3], Pz[3]); 
	glEnd();
	glColor3d(Red[4][0], Green[4][0], Blue[4][0]);
	glBegin(GL_LINE_STRIP);
    		glVertex3f(Px[4], Py[4], Pz[4]);
    		glVertex3f(Px[0], Py[0], Pz[0]);
	glEnd();
	
	//Drawing the floor
	glLineWidth(1.0);
	glColor3d(1.0,1.0,1.0);
	int floorSections = 100;
	float floorStartX = -5.0;
	float floorStopX = 5.0;
	float dx = (floorStopX - floorStartX)/floorSections;
	float floorStartZ = -5.0;
	float floorStopZ = 5.0;
	float dz = (floorStopZ - floorStartZ)/floorSections;
	float x;
	float z;
	
	x = floorStartX;
	for(i = 0; i < floorSections; i++)
	{
		glBegin(GL_LINE_STRIP);
			glVertex3f(x, 0.0, floorStartZ);   
			glVertex3f(x, 0.0, floorStopX); 
		glEnd();
		x += dx;
	}
	
	z = floorStartZ;
	for(i = 0; i < floorSections; i++)
	{
		glBegin(GL_LINE_STRIP);
			glVertex3f(floorStartX, 0.0, z);   
			glVertex3f(floorStopX, 0.0, z); 
		glEnd();
		z += dz;
	}
	
	//Pushing picture to the screen
	glutSwapBuffers();
}

float get_force(int i, int j, float separation)
{
	if(separation <= NaturalLength[i][j])
	{
		Red[i][j] = 1.0;
		Green[i][j] = 0.0;
		Blue[i][j] = 0.0;
		return(CompressionStrength[i][j]*(separation - NaturalLength[i][j]));
	}
	else
	{
		Red[i][j] = 0.0;
		Green[i][j] = 0.0;
		Blue[i][j] = 1.0;
		return(TensionStrength[i][j]*(separation - NaturalLength[i][j]));
	}
}

void n_body()
{
	float force_mag; 
	float dx,dy,dz,d, d2, dt;
	int    tdraw = 0; 
	int    tprint = 0;
	float  time = 0.0;
	int i,j;
	
	dt = DT;

	while(time < STOP_TIME)
	{
		for(i = 0; i < N; i++)
		{
			Fx[i] = 0.0;
			Fy[i] = 0.0;
			Fz[i] = 0.0;
		}
		
		for(i = 0; i < N; i++)
		{
			for(j = i+1; j < N; j++)
			{
				//Finding the distance between nodes.
				dx = Px[j] - Px[i];
				dy = Py[j] - Py[i];
				dz = Pz[j] - Pz[i];
				d2 = dx*dx + dy*dy + dz*dz;
				d  = sqrt(d2);
				
				//Getting the magnitude of the force caused by node positions.
				force_mag  =  get_force(i, j, d);
				
				//Seperating into x, y, z components 
				Fx[i] += force_mag*dx/d;
				Fx[j] -= force_mag*dx/d;
				Fy[i] += force_mag*dy/d;
				Fy[j] -= force_mag*dy/d;
				Fz[i] += force_mag*dz/d;
				Fz[j] -= force_mag*dz/d;
			}
			
			//Adding in the force of gravity
			Fy[i] += -G;
			
			//Adding in the push back force from the floor.
			if((Py[i] - SHERE_RADIUS) < 0.0) Fy[i] += FLOOR_STRENGTH*(0.0 - (Py[i] - SHERE_RADIUS)); 
		}

		//Leapfrog formulas to move the nodes forward in time dt.
		for(i = 0; i < N; i++)
		{
			if(time == 0.0)
			{
				Vx[i] += ((Fx[i]-DAMP*Vx[i])/Mass[i])*0.5*dt;
				Vy[i] += ((Fy[i]-DAMP*Vy[i])/Mass[i])*0.5*dt;
				Vz[i] += ((Fz[i]-DAMP*Vz[i])/Mass[i])*0.5*dt;
			}
			else
			{
				Vx[i] += ((Fx[i]-DAMP*Vx[i])/Mass[i])*dt;
				Vy[i] += ((Fy[i]-DAMP*Vy[i])/Mass[i])*dt;
				Vz[i] += ((Fz[i]-DAMP*Vz[i])/Mass[i])*dt;
			}

			Px[i] += Vx[i]*dt;
			Py[i] += Vy[i]*dt;
			Pz[i] += Vz[i]*dt;
		}

		if(tdraw == DRAW) 
		{
			draw_picture();
			tdraw = 0;
		}
		
		time += dt;
		tdraw++;
		tprint++;
	}
}

void control()
{	
	int    tdraw = 0;
	float  time = 0.0;
	set_initial_conditions();
	draw_picture();
    	n_body();
	
	printf("\n DONE \n");
	while(1);
}

void Display(void)
{
	gluLookAt(EYE, EYE, EYE, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, FAR);

	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Tower");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();

	return 0;
}
