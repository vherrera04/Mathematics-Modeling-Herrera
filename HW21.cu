#include "hip/hip_runtime.h"
//nvcc HW21.cu -o bounce2 -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BODIES 6
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BODIES], Velocity[NUMBER_OF_BODIES], Force[NUMBER_OF_BODIES], Color[NUMBER_OF_BODIES];
float BodyMass[NUMBER_OF_BODIES], BodyRadius[NUMBER_OF_BODIES];

int Trace;
int Pause;
int PrintRate;
int PrintCount;
double TotalBodyDistance;
double PrintBodyDistance;
int PolyCount, OctCount, OtherCount;
int Iteration, CheckCount;
int ViewBodies;

double SetupMaxVelocity;
double SetupGlobeSize;
double Damp;
double StopTolerance;
int CheckRate;
double DepletionForce;
double CentralForce;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitialConditions();
void setupBodies();
void drawPicture();
float4 centerOfMass();
void zeroOutSystem();
void getForces();
void updatePositions();
void nBody();
void startMeUp();
void terminalPrint();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'v')
	{
		if(ViewBodies == 1) ViewBodies = 0;
		else ViewBodies = 1;
		PrintCount = 0;
	}
	
	float dz = 0.05f;
	if(key == 'e')
	{
		glTranslatef(0.0, 0.0, -dz);
		drawPicture();
		terminalPrint();
	}
	if(key == 'E')
	{
		glTranslatef(0.0, 0.0, dz);
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'x') // Counter clockwise x-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(dAngle)*Position[i].y - sin(dAngle)*Position[i].z;
			Position[i].z  = sin(dAngle)*Position[i].y + cos(dAngle)*Position[i].z;
			Position[i].y  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	if(key == 'X') // Clockwise x-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(-dAngle)*Position[i].y - sin(-dAngle)*Position[i].z;
			Position[i].z  = sin(-dAngle)*Position[i].y + cos(-dAngle)*Position[i].z;
			Position[i].y  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	if(key == 'y') // Counter clockwise y-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(-dAngle)*Position[i].x - sin(-dAngle)*Position[i].z;
			Position[i].z  = sin(-dAngle)*Position[i].x + cos(-dAngle)*Position[i].z;
			Position[i].x  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	if(key == 'Y') // Clockwise y-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(dAngle)*Position[i].x - sin(dAngle)*Position[i].z;
			Position[i].z  = sin(dAngle)*Position[i].x + cos(dAngle)*Position[i].z;
			Position[i].x  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	if(key == 'z') // Counter clockwise y-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(dAngle)*Position[i].x - sin(dAngle)*Position[i].y;
			Position[i].y  = sin(dAngle)*Position[i].x + cos(dAngle)*Position[i].y;
			Position[i].x  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	if(key == 'Z') // Clockwise y-axis
	{
		float4 com = centerOfMass();
		float dAngle = 0.01;
		float temp;
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Position[i].x -= com.x;
			Position[i].y -= com.y;
			Position[i].z -= com.z;
			temp = cos(-dAngle)*Position[i].x - sin(-dAngle)*Position[i].y;
			Position[i].y  = sin(-dAngle)*Position[i].x + cos(-dAngle)*Position[i].y;
			Position[i].x  = temp;
			Position[i].x += com.x;
			Position[i].y += com.y;
			Position[i].z += com.z;
		}
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		printf("\nExiting....\n\nGood Bye\n");
		exit(0);
	}
}

void setInitialConditions()
{
	time_t t;
 	// Seeding the random number generater.
	srand((unsigned) time(&t));
 	
	double diameterOfPolystyrene, densityOfPolystyrene, volumeOfPolystyrene, massOfPolystyrene;
	//double Kb = (8.649828e-13); //km^3/kg*hr^2
	
	// The units that we will use to connect us to the outside world are: 
	// micrometers (um) 10^-6 meters
	// picograms (pg) 10^-12 grams
	// littleseconds (ls) 10^-4 seconds
	
	diameterOfPolystyrene = 1.0; // micrometer
	densityOfPolystyrene = 1.05; // g/cm^3
	densityOfPolystyrene *= 1e12/(1e4*1e4*1e4); //pg/(um^3) This does nothing just put it here for clarity.
	volumeOfPolystyrene = (PI/6.0)*diameterOfPolystyrene*diameterOfPolystyrene*diameterOfPolystyrene; // um^3
	massOfPolystyrene = volumeOfPolystyrene*densityOfPolystyrene;
	
	printf("\n Diameter of Polystyrene = %e micrometers", diameterOfPolystyrene);
	printf("\n Density of Polystyrene = %e picograms/micron^3", densityOfPolystyrene);
	printf("\n Volume of Polystyrene = %e micrometers^3", volumeOfPolystyrene);
	printf("\n Mass of Polystyrene = %e picograms", massOfPolystyrene);
	printf("\n");
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		BodyMass[i] = massOfPolystyrene;
		BodyRadius[i] = diameterOfPolystyrene/2.0;
		
		// Color of each body. 
		Color[i].x = 0.35;
		Color[i].y = 0.22;
		Color[i].z = 0.16;
	}
	printf("\n");
	
	setupBodies();
	
	// Making it run for 1 second.
	TotalRunTime = 10000.0;
	RunTime = 0.0;
	Dt = 0.001;
	
	printf("\n");
	printf("\n Units Have Been Set.");
	printf("\n");
}

void setupBodies()
{
	double seperation;
	int test;
	int tryCount;
	double angle1, angle2, radius;
	
	TotalBodyDistance = 10000000.0; // Just set it to a big number so if will fail the first test.
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// Settting the bodies randomly in a large sphere and not letting them be right on top of each other.
		test = 0;
		tryCount = 0;
		while(test == 0)
		{
			// Get random position.
			angle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			angle2 = ((float)rand()/(float)RAND_MAX)*PI;
			radius = ((float)rand()/(float)RAND_MAX)*SetupGlobeSize;
			Position[i].x = radius*cos(angle1)*sin(angle2);
			Position[i].y = radius*sin(angle1)*sin(angle2);
			Position[i].z = radius*cos(angle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < (BodyRadius[i] + BodyRadius[j]))
				{
					test = 0;
					break;
				}
			}
			tryCount++;
			if(1000 < tryCount)
			{
				printf("\n\n We tried 1000 times to set the position of body %d unsuccessfully.",i);
				printf("\n Something is wrong with your setup.");
				printf("\n Good Bye. \n");
				exit(0);
			}
		}
		
		// Setting random velocities between -SetupMaxVelocity and SetupMaxVelocity.
		Velocity[i].x = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*SetupMaxVelocity;
		Velocity[i].y = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*SetupMaxVelocity;
		Velocity[i].z = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*SetupMaxVelocity;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	// Drawing bodies.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(BodyRadius[i], 30, 30);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
    		centerOfMass.x += Position[i].x*BodyMass[i];
		centerOfMass.y += Position[i].y*BodyMass[i];
		centerOfMass.z += Position[i].z*BodyMass[i];
		totalMass += BodyMass[i];
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

void getForces()
{
	double inOut;
	double kSphere,kSphereReduction;
	float4 d, unit, dv;
	double magnitude;
	double intersectionArea; 
	double epsilon = 0.01;
	double r1,r2,temp;
	
	// Zeroing forces outside of the force loop just to be safe.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	kSphere = 10000.0;
	kSphereReduction = 0.2;
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{	
		// This adds forces between bodies.
		for(int j = 0; j < i; j++)
		{
			d.x = Position[j].x - Position[i].x;
			d.y = Position[j].y - Position[i].y;
			d.z = Position[j].z - Position[i].z;
			d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
			unit.x = d.x/d.w;
			unit.y = d.y/d.w;
			unit.z = d.z/d.w;
			
			// Nonelastic sphere collisions 
			if(d.w < (BodyRadius[i] + BodyRadius[j]))
			{
				// If the seperation gets too small the sphers may go through each other.
				// If you are ok with that you do not need this if statement.
				if(d.w < epsilon)
				{
					printf("\n Spheres %d and %d got to close. Make your sphere repultion stronger\n", i, j);
					exit(0);
				}
				
				// Finding which body is largest.
				if(BodyRadius[j] < BodyRadius[i])
				{
					r1 = BodyRadius[i];
					r2 = BodyRadius[j];
				}
				else
				{
					r1 = BodyRadius[j];
					r2 = BodyRadius[i];
				}
				
				// Finding the intection area.
				// The intersection area gets too large (If one spherez goes into the other).
				// Set it as the radius of the smaller sphere.
				temp = ((r2*r2 - r1*r1 + d.w*d.w)/(2.0*d.w));
				if(0.0 < temp)
				{
					intersectionArea = PI*(r2*r2 - temp*temp);
				}
				else
				{
					intersectionArea = PI*(r2*r2);
				}
				
				dv.x = Velocity[j].x - Velocity[i].x;
				dv.y = Velocity[j].y - Velocity[i].y;
				dv.z = Velocity[j].z - Velocity[i].z;
				inOut = d.x*dv.x + d.y*dv.y + d.z*dv.z;
				if(inOut < 0.0) magnitude = kSphere*intersectionArea; // If inOut is negative the sphere are converging.
				else magnitude = kSphereReduction*kSphere*intersectionArea; // If inOut is positive the sphere are diverging.
				
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*unit.x;
				Force[i].y -= magnitude*unit.y;
				Force[i].z -= magnitude*unit.z;
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*unit.x;
				Force[j].y += magnitude*unit.y;
				Force[j].z += magnitude*unit.z;
			}
			else if(d.w < (BodyRadius[i] + BodyRadius[j]) + 0.08) 
			{
				// This adds the depletion force between bodies.
				Force[i].x += DepletionForce*unit.x;
				Force[i].y += DepletionForce*unit.y;
				Force[i].z += DepletionForce*unit.z;
				
				Force[j].x -= DepletionForce*unit.x;
				Force[j].y -= DepletionForce*unit.y;
				Force[j].z -= DepletionForce*unit.z;
			}
		}
		
		// This adds a small central atraction force as a fraction of the depletion force.
		d.x = Position[i].x;
		d.y = Position[i].y;
		d.z = Position[i].z;
		d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
		unit.x = d.x/d.w;
		unit.y = d.y/d.w;
		unit.z = d.z/d.w;
		Force[i].x += CentralForce*unit.x;
		Force[i].y += CentralForce*unit.y;
		Force[i].z += CentralForce*unit.z;
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/BodyMass[i] - Velocity[i].x*Damp)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/BodyMass[i] - Velocity[i].y*Damp)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/BodyMass[i] - Velocity[i].z*Damp)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/BodyMass[i] - Velocity[i].x*Damp)*Dt;
			Velocity[i].y += (Force[i].y/BodyMass[i] - Velocity[i].y*Damp)*Dt;
			Velocity[i].z += (Force[i].z/BodyMass[i] - Velocity[i].z*Damp)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	float4 d;
	double newTotalBodyDistance;
	getForces();
	updatePositions();
	
	RunTime += Dt;
	PrintCount++;
	CheckCount++;
	
	if(CheckRate < CheckCount)
	{
		newTotalBodyDistance = 0.0;
		for(int i = 0; i < NUMBER_OF_BODIES - 1; i++)
		{
			for(int j = i + 1; j < NUMBER_OF_BODIES; j++)
			{
				d.x = Position[j].x - Position[i].x;
				d.y = Position[j].y - Position[i].y;
				d.z = Position[j].z - Position[i].z;
				d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
				newTotalBodyDistance += d.w;
			}
		}
		if(fabs(newTotalBodyDistance - TotalBodyDistance) < StopTolerance)
		{
			if(16.8 < newTotalBodyDistance && newTotalBodyDistance < 17.2) PolyCount++;
			else if(16.0 < newTotalBodyDistance && newTotalBodyDistance < 16.4) OctCount++;
			else OtherCount++;
			if(OctCount != 0)
			{
				printf("\n %d: %d, %d, %d Ratio = %f -- Distance = %f \n", Iteration, PolyCount, OctCount, OtherCount, (float)PolyCount/(float)OctCount, newTotalBodyDistance);
			}
			else
			{
				printf("\n %d: %d, %d, %d -- Distance = %f \n", Iteration, PolyCount, OctCount, OtherCount, newTotalBodyDistance);
			}
			TotalBodyDistance = 0.0;
			drawPicture();
			setupBodies();
			CheckCount = 0;
			Iteration++;
		}
		else
		{
			TotalBodyDistance = newTotalBodyDistance;
			CheckCount = 0;
		}
	}
	
	if(ViewBodies == 1)
	{
		drawPicture();
		
		if(PrintCount == PrintRate)
		{
			PrintBodyDistance = 0.0;
			for(int i = 0; i < NUMBER_OF_BODIES - 1; i++)
			{
				for(int j = i + 1; j < NUMBER_OF_BODIES; j++)
				{
					d.x = Position[j].x - Position[i].x;
					d.y = Position[j].y - Position[i].y;
					d.z = Position[j].z - Position[i].z;
					d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
					PrintBodyDistance += d.w;
				}
			}
			terminalPrint();
			PrintCount = 0;
		}
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	ViewBodies = 1;
	
	PrintRate = 10;
	PrintCount = 0;
	
	PolyCount = 0; 
	OctCount = 0;
	OtherCount = 0;
	
	CheckCount = 0;
	Iteration = 1;
	
	DepletionForce = 2.07097375; //pg*microM/MyS^2
	
	// Choose different central force strengths (should be grounded to the depletion force) and damping 
	// to see if you can find break points in the ratio of oct to poly.
	// Should run for a few hundred interations. Maybe a 1000.
	CentralForce = -0.57*DepletionForce;
	Damp = 0.1;
	
	SetupMaxVelocity = 1.1;
	SetupGlobeSize = 20.0;
	StopTolerance = 0.0001;
	CheckRate = 10000;
	
	setInitialConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n");
	printf("\n X/x: Clockwise/Counter Clockwise Rotation X-axis");
	printf("\n Y/y: Clockwise/Counter Clockwise Rotation Y-axis");
	printf("\n Z/z: Clockwise/Counter Clockwise Rotation Z-axis");
	printf("\n E/e: Zoom In/Zoom Out");
	
	printf("\n");
	printf("\033[0m");
	printf("\n t: Trace on/off toggle --> ");
	printf(" Tracing is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;32mON\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mOFF\n" "\e[m");
	}
	
	printf("\033[0m");
	printf(" p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf(" v: Viewing on/off toggle = %d", ViewBodies);
	printf("\n");
	printf(" q: Terminates the simulation");
	
	// Print the time out in hours. TotalBodyDistance;
	printf("\n\n Time = %f seconds 10^-4", RunTime);
	printf("\n TotalBodyDistance = %f microns", PrintBodyDistance);
	printf("\033[0m");
	printf("\n");
}

int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 50.0;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = SetupGlobeSize;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
