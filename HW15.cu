#include "hip/hip_runtime.h"
//nvcc HW15.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BODIES 95
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BODIES], Velocity[NUMBER_OF_BODIES], Force[NUMBER_OF_BODIES], Color[NUMBER_OF_BODIES];
// ????? you will put your masses and radii in here.
float BodyMass[NUMBER_OF_BODIES], BodyRadius[NUMBER_OF_BODIES];
// You will need to get ride of these and replace them with the ones above.
float SphereMass;
float SphereDiameter;
float MaxVelocity;
int Trace;
int Pause;
int PrintRate;
int PrintCount;
int WallCount;

// Units and universal constants
double MassUnitConverter;
double LengthUnitConverter;
double TimeUnitConverter;
float GravityConstant;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitialConditions();
void drawPicture();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();
void getForces();
void updatePositions();
void nBody();
void startMeUp();
void terminalPrint();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	if(key == 'g')
	{
		Pause = 1;
		terminalPrint();
		zeroOutSystem();
		
		for(int i = 0; i < NUMBER_OF_BODIES; i++)
		{
			Velocity[i].x += 20.0;
			Velocity[i].y += 0.0;
			Velocity[i].z += 0.0;
		}
		drawPicture();
		printf("\n Asteroids are ready to go.");
		printf("\n Press 'p' to hire them into the wall with a velocity of %f kilometers per hour.\n", 20.0*LengthUnitConverter/TimeUnitConverter);
	}
	
	if(key == 'k')
	{
		Pause = 1;
		terminalPrint();
		zeroOutSystem();
		drawPicture();
		printf("\n The simulation has been zeroed out.\n");
	}
	
	if(key == '1')
	{
		float4 pos, vel;
		Pause = 1;
		terminalPrint();
		pos = centerOfMass();
		printf("\n Center of mass = (%f, %f, %f)\n", pos.x, pos.y, pos.z); 
		vel = linearVelocity();
		printf("\n Linear velocity = (%f, %f, %f)\n", vel.x, vel.y, vel.z);
	}
	
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	
	float dx = 0.05f;
	if(key == 'x')
	{
		glTranslatef(-dx, 0.0, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'X')
	{
		glTranslatef(dx, 0.0, 0.0);
		drawPicture();
	}
	
	float dy = 0.05f;
	if(key == 'y')
	{
		glTranslatef(0.0, -dy, 0.0);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Y')
	{
		glTranslatef(0.0, dy, 0.0);
		drawPicture();
	}
	
	float dz = 0.05f;
	if(key == 'z')
	{
		glTranslatef(0.0, 0.0, -dz);
		drawPicture();
		terminalPrint();
	}
	if(key == 'Z')
	{
		glTranslatef(0.0, 0.0, dz);
		drawPicture();
	}
	
	if(key == 'q')
	{
		glutDestroyWindow(Window);
		printf("\nExiting....\n\nGood Bye\n");
		exit(0);
	}
}

void setInitialConditions()
{
	time_t t;
	float seperation;
	int test;
	float globeSize, angle1, angle2, radius;
	double massOfCeres;
	double diameterOfCeres;
	double densityOfCeres;
	double G = (8.649828e-13); //km^3/kg*hr^2
	
	// Seeding the random number generater.
	srand((unsigned) time(&t));
	
	// The units that we will use to contect us to the outside world are: 
	// kilometers (km)
	// kilograms (kg)
	// hours (hr)
	// If you multiply one of our units by this number it will convert it the outside world units.
	// If you divide an outside world unit by this number it will convert it to our units
	// We are setting the mass unit to be the mass of Ceres.
	// We are settting the length unit to be th diameter of Ceres.
	// We are setting the time unit to be the such that the universal gravity constant is 1.
	
	massOfCeres = 9.383e20; // kg
	diameterOfCeres = 940.0; // km
	densityOfCeres = massOfCeres/((PI/6.0)*diameterOfCeres*diameterOfCeres*diameterOfCeres); // kg/km^3
	
	// ??????????
	// Use random numbers to get all your different mass bodies
	// BodyMass[i] = ????
	
	// ??? Set you mass unit
	MassUnitConverter = massOfCeres/NUMBER_OF_BODIES; // kg
	
	// From the random masses you just did set all your corresponding radii
	// BodyRadius[] = ???? 
	
	// Set your length unit
	LengthUnitConverter = pow((massOfCeres*6.0/(PI*NUMBER_OF_BODIES*densityOfCeres)),1.0/3.0); // km
	TimeUnitConverter = sqrt(LengthUnitConverter*LengthUnitConverter*LengthUnitConverter/(G*MassUnitConverter)); // hr
	
	printf("\n MassUnitConverter = %e kilograms", MassUnitConverter);
	printf("\n LengthUnitConverter = %e kilometers", LengthUnitConverter);
	printf("\n TimeUnitConverter = %e hours", TimeUnitConverter);
	
	// If we did everthing right the universal gravity constant should be 1.
	GravityConstant = 1.0;
	printf("\n The gravity constant = %f in our units", GravityConstant);
	
	// All spheres are the same diameter and mass so these should be 1. Noy true
	// ??? don't need these.They will be close to one but not exactly one anymore.
	// Also you will just need to replace all these in the code with your new ones.
	// Have fun finding them all!!!
	SphereDiameter = 1.0;
	SphereMass = 1.0;
	
	// Making the size of the intial globe we use to place the bodies.
	globeSize = 10.0*SphereDiameter;
	
	// You get to pick this but it is nice to print it out in common units to get a feel for what it is.
	MaxVelocity = 1.0;
	printf("\n Max velocity = %f kilometers/hour or %f miles/hour", MaxVelocity*LengthUnitConverter/TimeUnitConverter, (MaxVelocity*LengthUnitConverter/TimeUnitConverter)*0.621371);
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// Settting the balls randomly in a large sphere and not letting them be right on top of each other.
		test = 0;
		while(test == 0)
		{
			// Get random position.
			angle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			angle2 = ((float)rand()/(float)RAND_MAX)*PI;
			radius = ((float)rand()/(float)RAND_MAX)*globeSize;
			Position[i].x = radius*cos(angle1)*sin(angle2);
			Position[i].y = radius*sin(angle1)*sin(angle2);
			Position[i].z = radius*cos(angle2);
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < SphereDiameter)
				{
					test = 0;
					break;
				}
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		Velocity[i].x = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		
		// Color of each asteroid. 
		Color[i].x = 0.35;
		Color[i].y = 0.22;
		Color[i].z = 0.16;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	// Making it run for 10 days.
	// Taking days to hours then to our units.
	TotalRunTime = 10.0*24.0/TimeUnitConverter;
	RunTime = 0.0;
	Dt = 0.001;
	// How many time steps between termenal prints
	PrintRate = 10;
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(SphereDiameter/2.0, 30, 30);
		glPopMatrix();
	}
	
	// Drawing the wall.
	glColor3d(1.0, 1.0, 0.75);
	glBegin(GL_QUADS);
		glVertex3f(25.0, -5.0, 5.0);
		glVertex3f(25.0, -5.0, -5.0);
		glVertex3f(25.0, 5.0, -5.0);
		glVertex3f(25.0, 5.0, 5.0);
		glVertex3f(25.0, -5.0, 5.0);
	glEnd();
	WallCount++;
	
	glColor3d(1.0, 0.0, 0.0);
	glPointSize(10.0f);
	glBegin(GL_POINTS);
		glVertex3f(25.0f, 0.0f, 0.0f);
	glEnd();
	
	glutSwapBuffers();
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
    		centerOfMass.x += Position[i].x*SphereMass;
		centerOfMass.y += Position[i].y*SphereMass;
		centerOfMass.z += Position[i].z*SphereMass;
		totalMass += SphereMass;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
    		linearVelocity.x += Velocity[i].x*SphereMass;
		linearVelocity.y += Velocity[i].y*SphereMass;
		linearVelocity.z += Velocity[i].z*SphereMass;
		totalMass += SphereMass;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		Position[i].x -= pos.x;
		Position[i].y -= pos.y;
		Position[i].z -= pos.z;
		
		Velocity[i].x -= vel.x;
		Velocity[i].y -= vel.y;
		Velocity[i].z -= vel.z;
	}
}

void getForces()
{
	float inOut;
	float kSphere,kSphereReduction;
	float kWall, kWallReduction;
	float4 d, unit, dv;
	float magnitude;
	float intersectionArea; 
	float sphereRadius = SphereDiameter/2.0;
	
	// Zeroing forces outside of the force loop just to be safe.
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	kWall = 20000.0;
	kWallReduction = 0.2;
	kSphere = 10000.0;
	kSphereReduction = 0.5;
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{	
		if(25.0 < Position[i].x + SphereDiameter/2.0 && Position[i].x + SphereDiameter/2.0 < 26.0)
		{
			if(-5.0 < Position[i].z && Position[i].z < 5.0 && -5.0 < Position[i].z && Position[i].z < 5.0)
			{
				if(0.0 < Velocity[i].x)
				{
					magnitude = (Position[i].x + SphereDiameter/2.0 - 25.0)*kWall;
				}
				else
				{
					magnitude = (Position[i].x + SphereDiameter/2.0 - 25.0)*kWall*kWallReduction;
				}
				Force[i].x -= magnitude;
			}
		}
		
		
		// This adds forces between asteriods.
		for(int j = 0; j < i; j++)
		{
			d.x = Position[j].x - Position[i].x;
			d.y = Position[j].y - Position[i].y;
			d.z = Position[j].z - Position[i].z;
			d.w = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
			unit.x = d.x/d.w;
			unit.y = d.y/d.w;
			unit.z = d.z/d.w;
			
			// Nonelastic sphere collisions 
			if(d.w < SphereDiameter)
			{
				// If the seperation gets too small the sphers may go through each other.
				// If you are ok with that you do not need this line.
				if(d.w < sphereRadius/10.0)
				{
					printf("\n Spheres %d and %d got to close. Make your sphere repultion stronger\n", i, j);
					exit(0);
				}
				
				intersectionArea = (PI/4.0)*(SphereDiameter*SphereDiameter - d.w*d.w);
				
				dv.x = Velocity[j].x - Velocity[i].x;
				dv.y = Velocity[j].y - Velocity[i].y;
				dv.z = Velocity[j].z - Velocity[i].z;
				inOut = d.x*dv.x + d.y*dv.y + d.z*dv.z;
				if(inOut < 0.0) magnitude = kSphere*intersectionArea; // If inOut is negative the sphere are converging.
				else magnitude = kSphereReduction*kSphere*intersectionArea; // If inOut is positive the sphere are diverging.
				
				//if(inOut < 0.0) magnitude = kSphere*(SphereDiameter - d.w); // If inOut is negative the sphere are converging.
				//else magnitude = kSphereReduction*kSphere*(SphereDiameter - d.w); // If inOut is positive the sphere are diverging.
				
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*unit.x;
				Force[i].y -= magnitude*unit.y;
				Force[i].z -= magnitude*unit.z;
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*unit.x;
				Force[j].y += magnitude*unit.y;
				Force[j].z += magnitude*unit.z;
				
				// This adds the gravity between asteroids but the gravity is lock in at what it 
				// was at impact.
				magnitude = GravityConstant*SphereMass*SphereMass/(SphereDiameter*SphereDiameter);
				Force[i].x += magnitude*unit.x;
				Force[i].y += magnitude*unit.y;
				Force[i].z += magnitude*unit.z;
				
				Force[j].x -= magnitude*unit.x;
				Force[j].y -= magnitude*unit.y;
				Force[j].z -= magnitude*unit.z;
			}
			else
			{
				// This adds the gravity between asteroids when they are not touching.
				magnitude = GravityConstant*SphereMass*SphereMass/(d.w*d.w);
				Force[i].x += magnitude*unit.x;
				Force[i].y += magnitude*unit.y;
				Force[i].z += magnitude*unit.z;
				
				Force[j].x -= magnitude*unit.x;
				Force[j].y -= magnitude*unit.y;
				Force[j].z -= magnitude*unit.z;
			}
		}
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BODIES; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/SphereMass)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/SphereMass)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/SphereMass)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/SphereMass)*Dt;
			Velocity[i].y += (Force[i].y/SphereMass)*Dt;
			Velocity[i].z += (Force[i].z/SphereMass)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	
	RunTime += Dt;
	PrintCount++;
	
	if(PrintCount == PrintRate)
	{
		terminalPrint();
		PrintCount = 0;
	}
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	PrintCount = 0;
	setInitialConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n");
	printf("\n X/x: Move Right move left");
	printf("\n Y/y: Move Up move down");
	printf("\n Z/z: Move in move out");
	
	printf("\n");
	printf("\n k: Will zero out the center of mass and linear velocity of the system.");
	printf("\n 1: Will print the center of mass and the linear velocity of the system.");
	printf("\n");
	printf("\n g: Will center the asteroids then fire them into the wall on your comand.");
	printf("\n");
	printf("\033[0m");
	printf("\n t: Trace on/off toggle --> ");
	printf(" Tracing is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;32mON\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mOFF\n" "\e[m");
	}
	
	printf("\033[0m");
	printf(" p: pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf(" q: Terminates the simulation");
	
	// Print the time out in hours.
	printf("\n\n Time = %f \033[0;34mhours", RunTime*TimeUnitConverter);
	printf("\033[0m");
	printf("\n");
}


int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 50.0*SphereDiameter;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 25.0*SphereDiameter;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
