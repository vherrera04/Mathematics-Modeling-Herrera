#include "hip/hip_runtime.h"
//nvcc HW3.cu -o myHW3 -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BALLS 20
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BALLS], Velocity[NUMBER_OF_BALLS], Force[NUMBER_OF_BALLS], Color[NUMBER_OF_BALLS];
float SphereMass;
float SphereDiameter;
float BoxSideLength;
float MaxVelocity;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

void setInitailConditions();
void drawPicture();
void getForces();
void updatePositions();
void handleCollisions();
void nBody();
void startMeUp();

void Display()
{
	drawPicture();
}

void idle()
{
	nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void setInitailConditions()
{
	time_t t;
	float randomNumber;
	float halfBoxSideLength;
	float sphereRadius;
	float seperation;
	int test;
	
	// Seading the random number generater.
	srand((unsigned) time(&t));
	
	SphereDiameter = 0.5;
	sphereRadius = SphereDiameter/2.0;
	SphereMass = 1.0;
	BoxSideLength = 5.0;
	MaxVelocity = 10.0;
	halfBoxSideLength = BoxSideLength/2.0;
	
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// Settting the balls randomly in the box and not letting them be right on top of each other.
		test = 0;
		while(test == 0)
		{
			// Get random 0 and 1.
			randomNumber = ((float)rand()/(float)RAND_MAX);
			// Making it between -1 and 1
			randomNumber = randomNumber*2.0 - 1.0;
			// Making it between -halfBoxSideLength and halfBoxSideLength
			randomNumber = randomNumber*(halfBoxSideLength - sphereRadius);
			// Putting the ball there x.
			Position[i].x = randomNumber;
			
			// Get random 0 and 1.
			randomNumber = ((float)rand()/(float)RAND_MAX);
			// Making it between -1 and 1
			randomNumber = randomNumber*2.0 - 1.0;
			// Making it between -halfBoxSideLength and halfBoxSideLength
			randomNumber = randomNumber*(halfBoxSideLength - sphereRadius);
			// Putting the ball there y.
			Position[i].y = randomNumber;
			
			// Get random 0 and 1.
			randomNumber = ((float)rand()/(float)RAND_MAX);
			// Making it between -1 and 1
			randomNumber = randomNumber*2.0 - 1.0;
			// Making it between -halfBoxSideLength and halfBoxSideLength
			randomNumber = randomNumber*(halfBoxSideLength - sphereRadius);
			// Putting the ball there z.
			Position[i].z = randomNumber;
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these position away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < SphereDiameter)
				{
					test = 0;
					break;
				}
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].x = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = randomNumber;
		
		// Randomly coloring the balls
		randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].x = randomNumber;
		randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].y = randomNumber;
		randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].z = randomNumber;
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	TotalRunTime = 10000.0;
	RunTime = 0.0;
	Dt = 0.001;
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	float halfSide = BoxSideLength/2.0;
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(SphereDiameter/2.0, 30, 30);
		glPopMatrix();
	}
	
	glLineWidth(3.0);
	//Drawing front of box
	glColor3d(0.0, 1.0, 0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
	glEnd();
	//Drawing back of box
	glColor3d(1.0, 1.0, 1.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off right side
	glBegin(GL_LINES);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off left side
	glBegin(GL_LINES);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	
	
	glutSwapBuffers();
}

void getForces()
{
	float wallStiffnessIn = 10000.0;
	float wallStiffnessOut = 8000.0;
	float k;
	float halfSide = BoxSideLength/2.0;
	float howMuch;
	float ballRadius = SphereDiameter/2.0;
	
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
		
		if((Position[i].x - ballRadius) < -halfSide)
		{
			howMuch = -halfSide - (Position[i].x - ballRadius);
			if(Velocity[i].x < 0.0) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].x += k*howMuch;
		}
		else if(halfSide < (Position[i].x + ballRadius))
		{
			howMuch = (Position[i].x + ballRadius) - halfSide;
			if(0.0 < Velocity[i].x) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].x -= k*howMuch;
		}
		
		if((Position[i].y - ballRadius) < -halfSide)
		{
			howMuch = -halfSide - (Position[i].y - ballRadius);
			if(Velocity[i].y < 0.0) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].y += k*howMuch;
		}
		else if(halfSide < (Position[i].y + ballRadius))
		{
			howMuch = (Position[i].y + ballRadius) - halfSide;
			if(0.0 < Velocity[i].y) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].y -= k*howMuch;
		}
		
		if((Position[i].z - ballRadius) < -halfSide)
		{
			howMuch = -halfSide - (Position[i].z - ballRadius);
			if(Velocity[i].z < 0.0) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].z += k*howMuch;
		}
		else if(halfSide < (Position[i].z + ballRadius))
		{
			howMuch = (Position[i].z + ballRadius) - halfSide;
			if(0.0 < Velocity[i].z) k = wallStiffnessIn;
			else k = wallStiffnessOut;
			Force[i].z -= k*howMuch;
		}
	}
}

void handleCollisions()
{
    float distance;
    float normalX, normalY, normalZ;
    float relativeVelocityX, relativeVelocityY, relativeVelocityZ;
    float dotProduct;
    
    for(int i = 0; i < NUMBER_OF_BALLS; i++)
    {
        for(int j = i + 1; j < NUMBER_OF_BALLS; j++)
        {
            // Calculate distance between ball centers
            distance = sqrt((Position[i].x - Position[j].x) * (Position[i].x - Position[j].x) +
                            (Position[i].y - Position[j].y) * (Position[i].y - Position[j].y) +
                            (Position[i].z - Position[j].z) * (Position[i].z - Position[j].z));
            
            if(distance < SphereDiameter)
            {
                // Normal vector from i to j
                normalX = (Position[j].x - Position[i].x) / distance;
                normalY = (Position[j].y - Position[i].y) / distance;
                normalZ = (Position[j].z - Position[i].z) / distance;
                
                // Relative velocity
                relativeVelocityX = Velocity[j].x - Velocity[i].x;
                relativeVelocityY = Velocity[j].y - Velocity[i].y;
                relativeVelocityZ = Velocity[j].z - Velocity[i].z;
                
                // Dot product
                dotProduct = normalX * relativeVelocityX +
                             normalY * relativeVelocityY +
                             normalZ * relativeVelocityZ;
                
                // Calculate new velocities
                float massSum = SphereMass + SphereMass;
                float coefficient = dotProduct / massSum;
                
                Velocity[i].x += coefficient * normalX * SphereMass;
                Velocity[i].y += coefficient * normalY * SphereMass;
                Velocity[i].z += coefficient * normalZ * SphereMass;
                
                Velocity[j].x -= coefficient * normalX * SphereMass;
                Velocity[j].y -= coefficient * normalY * SphereMass;
                Velocity[j].z -= coefficient * normalZ * SphereMass;
                
                // Update positions to ensure they are not intersecting
                float overlap = SphereDiameter - distance;
                Position[i].x -= normalX * overlap / 2.0;
                Position[i].y -= normalY * overlap / 2.0;
                Position[i].z -= normalZ * overlap / 2.0;
                
                Position[j].x += normalX * overlap / 2.0;
                Position[j].y += normalY * overlap / 2.0;
                Position[j].z += normalZ * overlap / 2.0;
            }
        }
    }
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/SphereMass)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/SphereMass)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/SphereMass)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/SphereMass)*Dt;
			Velocity[i].y += (Force[i].y/SphereMass)*Dt;
			Velocity[i].z += (Force[i].z/SphereMass)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	getForces();
	handleCollisions();
	updatePositions();
	drawPicture();
	printf("\n Time = %f", RunTime);
	RunTime += Dt;
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	setInitailConditions();
}

int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	//Far = BoxSideLength;
	Far = 10.0;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 6.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	//glutMouseFunc(mymouse);
	//glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
