#include "hip/hip_runtime.h"
//nvcc HW5.cu -o bounce -lglut -lm -lGLU -lGL																													
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER_OF_BALLS 20
#define PI 3.14159
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position[NUMBER_OF_BALLS], Velocity[NUMBER_OF_BALLS], Force[NUMBER_OF_BALLS], Color[NUMBER_OF_BALLS];
float SphereMass;
float SphereDiameter;
float BoxSideLength;
float MaxVelocity;
int Trace;
int Pause;
// ????????????????????????????????????
// I did this for you you just need to fill them in later.
float MassUnitConverter;
float LengthUnitConverter;
float TimeUnitConverter;
float GravityConstant;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void Display();
void idle();
void reshape(int, int);
void KeyPressed(unsigned char, int, int);
void setInitailConditions();
void drawPicture();
void getForces();
void updatePositions();
void nBody();
void startMeUp();

void Display()
{
	drawPicture();
}

void idle()
{
	if(Pause == 0) nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{
	// Turns tracers on and off
	if(key == 't')
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
	}
	
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
	}
}

void setInitailConditions()
{
	time_t t;
	float randomNumber;
	float halfBoxSideLength;
	float sphereRadius;
	float seperation;
	int test;
	
	// Seeding the random number generater.
	srand((unsigned) time(&t));
	
	// ??????????????????????????????????????????????????????????
	// For the units that we will use to connect us to the outside world let use 
	// kilometers (km)
	// kilograms (kg)
	// hours (hr)
	// If you multiply one of our units by this number it will convert it to the outside world units.
	// If you divide an outside world unit by this number it will convert it to our units
	// Set your conversion units then print them out.
	// Uncomment these and fix them.
	MassUnitConverter = 9.383e20 ; // kg From HW4
	LengthUnitConverter = 940.0; // km From HW4
	TimeUnitConverter = 3640.0 / 3600.0; // hr From HW4
	printf("\n MassUnitConverter = %f kilograms", MassUnitConverter);
	printf("\n LengthUnitConverter = %f kilometers", LengthUnitConverter);
	printf("\n TimeUnitConverter = %f hours", TimeUnitConverter);
	
	// ??????????????????????????????????????????????????????????
	// Set the GravityConstant. and print it out.
	// Uncomment these and fix them.
	GravityConstant = 1.0; 
	printf("\n The gravity constant = %f in our units", GravityConstant);
	
	// ??????????????????????????????????????????????????????????
	// Anything with a mass, time or length needs to be thought about.
	// Comment about each of these. Most will may not need to be changed but just say why.
	SphereDiameter = 0.5;
	sphereRadius = SphereDiameter/2.0;
	SphereMass = 1.0;
	BoxSideLength = 5.0;
	MaxVelocity = 10.0;
	halfBoxSideLength = BoxSideLength/2.0;

	// ??????????????????????????????????????????????????????????
	// Print out how many kilometers long each box side is.
	// Print out how many kilometers/hour the max Velocity is.
	// Uncomment these and fix them.
	printf("\n Box side length = %f kilometers", BoxSideLength);
	printf("\n Max velocity = %f kilometers/hour", MaxVelocity);
	
	
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// Setting the balls randomly in the box and not letting them be right on top of each other.
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*(halfBoxSideLength - sphereRadius);
			Position[i].x = randomNumber;
			randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*(halfBoxSideLength - sphereRadius);
			Position[i].y = randomNumber;
			randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*(halfBoxSideLength - sphereRadius);
			Position[i].z = randomNumber;
			
			// Making sure the balls centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = sqrt((Position[i].x-Position[j].x)*(Position[i].x-Position[j].x) + (Position[i].y-Position[j].y)*(Position[i].y-Position[j].y) + (Position[i].z-Position[j].z)*(Position[i].z-Position[j].z));
				if(seperation < SphereDiameter)
				{
					test = 0;
					break;
				}
			}
		}
		
		// Setting random velocities between -MaxVelocity and MaxVelocity.
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].x = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].y = randomNumber;
		randomNumber = (((float)rand()/(float)RAND_MAX)*2.0 - 1.0)*MaxVelocity;
		Velocity[i].z = randomNumber;

		// ?????????????????????????????????????????
		// Asteriods are brown not just any color. 
		// Well I have not seen many asteriods maybe they are all the colors in the rainbow.
		// But make them brown anyway. 
		//randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].x = 0.36 ;//red
		//randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].y = 0.25 ;//green
		//randomNumber = ((float)rand()/(float)RAND_MAX);
		Color[i].z = 0.2 ;//blue
		
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;
	}
	
	// ?????????????????????????????????????????
	// Make this a 10 day long run
	TotalRunTime = 864000.0; // convert to seconds by multiplying 10 days by 24 hours. multiply that by 3600 seconds
	RunTime = 0.0;
	Dt = 0.001;
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
	
	float halfSide = BoxSideLength/2.0;
	
	// Drawing balls.
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		glColor3d(Color[i].x, Color[i].y, Color[i].z);
		glPushMatrix();
			glTranslatef(Position[i].x, Position[i].y, Position[i].z);
			glutSolidSphere(SphereDiameter/2.0, 30, 30);
		glPopMatrix();
	}
	
	glLineWidth(3.0);
	//Drawing front of box
	glColor3d(0.0, 1.0, 0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
	glEnd();
	//Drawing back of box
	glColor3d(1.0, 1.0, 1.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off right side
	glBegin(GL_LINES);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off left side
	glBegin(GL_LINES);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	
	glutSwapBuffers();
}

void getForces()
{
	float wallStiffnessIn = 10000.0;
	float wallStiffnessOut = 8000.0;
	float kWall, kBall;
	float halfSide = BoxSideLength/2.0;
	float amountOut;
	float ballRadius = SphereDiameter/2.0;
	float d, dx, dy, dz;
	float magnitude;
	
	kBall = 1000.0;
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		Force[i].x = 0.0;
		Force[i].y = 0.0;
		Force[i].z = 0.0;

		if((Position[i].x - ballRadius) < -halfSide)
		{
			amountOut = -halfSide - (Position[i].x - ballRadius);
			if(Velocity[i].x < 0.0) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].x += kWall*amountOut;
		}
		else if(halfSide < (Position[i].x + ballRadius))
		{
			amountOut = (Position[i].x + ballRadius) - halfSide;
			if(0.0 < Velocity[i].x) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].x -= kWall*amountOut;
		}
		
		if((Position[i].y - ballRadius) < -halfSide)
		{
			amountOut = -halfSide - (Position[i].y - ballRadius);
			if(Velocity[i].y < 0.0) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].y += kWall*amountOut;
		}
		else if(halfSide < (Position[i].y + ballRadius))
		{
			amountOut = (Position[i].y + ballRadius) - halfSide;
			if(0.0 < Velocity[i].y) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].y -= kWall*amountOut;
		}
		
		if((Position[i].z - ballRadius) < -halfSide)
		{
			amountOut = -halfSide - (Position[i].z - ballRadius);
			if(Velocity[i].z < 0.0) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].z += kWall*amountOut;
		}
		else if(halfSide < (Position[i].z + ballRadius))
		{
			amountOut = (Position[i].z + ballRadius) - halfSide;
			if(0.0 < Velocity[i].z) kWall = wallStiffnessIn;
			else kWall = wallStiffnessOut;
			Force[i].z -= kWall*amountOut;
		}
		
		for(int j = 0; j < i; j++)
		{
			dx = Position[j].x - Position[i].x;
			dy = Position[j].y - Position[i].y;
			dz = Position[j].z - Position[i].z;
			d = sqrt(dx*dx + dy*dy + dz*dz);
			
			// This causes the asteroids to bounce off of each other.
			if(d < SphereDiameter)  
			{
				magnitude = kBall*(SphereDiameter - d);
				// Doling out the force in the proper perfortions using unit vectors.
				Force[i].x -= magnitude*(dx/d);
				Force[i].y -= magnitude*(dy/d);
				Force[i].z -= magnitude*(dz/d);
				// A force on me causes the opposite force on you. 
				Force[j].x += magnitude*(dx/d);
				Force[j].y += magnitude*(dy/d);
				Force[j].z += magnitude*(dz/d);
			}
			
			// ???????????????????????????????????????????????????????
			// Add gravity between asteroids here.
			if (d > 0.0) // This is to prevent division by zero
            		{
              			float gravitationalForceMagnitude = GravityConstant * SphereMass * SphereMass / (d * d);
               			// Apply gravitational force in the direction of the other ball
                		Force[i].x += gravitationalForceMagnitude * (dx / d);
               		 	Force[i].y += gravitationalForceMagnitude * (dy / d);
                		Force[i].z += gravitationalForceMagnitude * (dz / d);
                
                		// Apply opposite gravitational force to the other ball
                		Force[j].x -= gravitationalForceMagnitude * (dx / d);
                		Force[j].y -= gravitationalForceMagnitude * (dy / d);
                		Force[j].z -= gravitationalForceMagnitude * (dz / d);
            		}

			
			// Two elderly ladies get pulled over by a cop on I-35 in Dallas.
			// The cop says "Mam you were going 35 miles an hour in a 70. You are causing a trafic jam 
			// and may get someone, perhaps yourself, hurt".
			// He turns his atention to the lady in the pasangers seat and says "mam
			// are you okay", because she was breathing really hard and looked completely freaked out.
			// She replied " Yes young man I will be okay in a minute. We just pulled off of 
			// highway 114.
			
		}
	}
}

void updatePositions()
{
	for(int i = 0; i < NUMBER_OF_BALLS; i++)
	{
		// These are the LeapFrog formulas.
		if(RunTime == 0.0)
		{
			Velocity[i].x += (Force[i].x/SphereMass)*(Dt/2.0);
			Velocity[i].y += (Force[i].y/SphereMass)*(Dt/2.0);
			Velocity[i].z += (Force[i].z/SphereMass)*(Dt/2.0);
		}
		else
		{
			Velocity[i].x += (Force[i].x/SphereMass)*Dt;
			Velocity[i].y += (Force[i].y/SphereMass)*Dt;
			Velocity[i].z += (Force[i].z/SphereMass)*Dt;
		}

		Position[i].x += Velocity[i].x*Dt;
		Position[i].y += Velocity[i].y*Dt;
		Position[i].z += Velocity[i].z*Dt;
	}
}

void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	// ??????????????????????????????????????????????
	// Print the time out in hours.
	double RunTimeInHours = RunTime / 24.0; //used to convert seconds into hours
	printf("\n Time = %f hours", RunTimeInHours);
	RunTime += Dt;
	
	if(RunTime >= TotalRunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	Trace = 0;
	Pause = 1;
	setInitailConditions();
	printf("\033[0;31m\n\n The simulation is paused. Type p in the simulation window to start it. \n");
	printf("\033[0m");
}

int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 
	
	// Clip plains
	Near = 0.2;
	Far = 2.2*BoxSideLength;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 1.1*BoxSideLength;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_Position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_Position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	//glutMouseFunc(mymouse);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
